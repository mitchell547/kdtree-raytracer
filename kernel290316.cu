// triangle.cpp: ���������� ����� ����� ��� ����������� ����������.
//
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <math.h>   
#include <stdlib.h> 
#include <random>
#include <iostream>
#include <omp.h>
#define EPSILON 0.000000001
#define INF 1e20
#define standartTest//baricentTest
#define bmp
//#define sampleInCenter
#define filter


struct Vec
{        // Usage: time ./smallpt 5000 && xv image.ppm 
	double x, y, z;                  // position, also color (r,g,b) 
	inline __host__ __device__ Vec (double x_ = 0, double y_ = 0, double z_ = 0)
	{
		x = x_;
		y = y_;
		z = z_;
	}
	inline __host__ __device__ Vec		operator+	  (const Vec &b) const
	{
		return Vec (x + b.x, y + b.y, z + b.z);
	}
	inline __host__ __device__ Vec		operator-	  (const Vec &b) const
	{
		return Vec (x - b.x, y - b.y, z - b.z);
	}
	inline __host__ __device__ Vec		operator*	  (double b)	 const
	{
		return Vec (x*b, y*b, z*b);
	}
	inline __host__ __device__ Vec		operator/	  (double b)	 const
	{
		return Vec (x/b, y/b, z/b);
	}
	inline __host__ __device__ Vec		inv			  ()			 const
	{
		return Vec (-x, -y, -z);
	}
	inline __host__ __device__ Vec		mult		  (const Vec &b) const
	{
		return Vec (x*b.x, y*b.y, z*b.z);
	}
	inline __host__ __device__ double	distance	  (const Vec &b) const
	{
		return  sqrt ((x - b.x)*(x - b.x) +
			(y - b.y)*(y - b.y) +
			(z - b.z)*(z - b.z));
	}
	inline __host__ __device__ Vec&		normalization ()
	{
		return *this = *this * (1 / sqrt (x*x + y*y + z*z));
	}
	inline __host__ __device__ double	dot			  (const Vec &b) const
	{
		return x*b.x + y*b.y + z*b.z;
	} // cross: 
	inline __host__ __device__ Vec		cross		  (const Vec&b)  const
	{
		return Vec (y*b.z - z*b.y, z*b.x - x*b.z, x*b.y - y*b.x);
	}
};
struct Ray
{
	Vec o, d; //o-������ d-�����������
	__host__ __device__ Ray (Vec o_, Vec d_) : o (o_), d (d_) {}
};
struct triangle
{
	Vec a1, a2, a3, c, _normal;      // position, color 
	Vec o; double rad;//spher
	double difra = 1; double reflect;
	inline __host__ __device__ triangle (Vec a1_, Vec a2_, Vec a3_, Vec c_, double reflect_) :
		a1 (a1_), a2 (a2_), a3 (a3_), c (c_), reflect (reflect_)
	{
		_normal = (a2 - a1).cross(a3 - a1);

		makeMinimumBoundingSphere (a1, a2, a3);
		
	}
	void  makeMinimumBoundingSphere (const Vec &p1, const Vec &p2, const Vec &p3) {
		
		// Calculate relative distances
		float A = (p1  ).distance (p2);
		float B = (p2  ).distance (p3);
		float C = (p3 ).distance (p1);

		// Re-orient triangle (make A longest side)
		const Vec *a = &p3, *b = &p1, *c = &p2;
		if (B < C) swap (B, C), swap (b, c);
		if (A < B) swap (A, B), swap (a, b);

		// If obtuse, just use longest diameter, otherwise circumscribe
		if ((B*B) + (C*C) <= (A*A)) {
			rad = A / 2.f;
			o = (*b + *c)/ 2.f;
		}
		else {
			// http://en.wikipedia.org/wiki/Circumscribed_circle
			float  cos_a = (B*B + C*C - A*A) / (B*C * 2);
			rad = A / (sqrt (1 - cos_a*cos_a)*2.f);
			Vec alpha = *a - *c, beta = *b - *c;
			o = (beta * alpha.dot (alpha) - alpha * beta.dot (beta)).cross (alpha.cross (beta)) /
				(alpha.cross (beta).dot (alpha.cross (beta)) * 2.f) + *c;
		}
		
	}
	void swap (float & A, float & B)
	{
		float C = A;
		A = B;
		B = C;
	}
	void swap (const Vec * A, const Vec * B)
	{
		const Vec * C = A;
		A = B;
		B = C;
	}
	inline __host__ __device__ triangle (){}

	inline __host__ __device__ triangle moveX (const double step)const
	{
		triangle newT = *this;
		newT.a1.x += step;
		newT.a2.x += step;
		newT.a3.x += step;
		return newT;
	}
	inline __host__ __device__ triangle moveY (const double step)const
	{
		triangle newT = *this;
		newT.a1.y += step;
		newT.a2.y += step;
		newT.a3.y += step;
		return newT;
	}
	inline __host__ __device__ triangle moveZ (const double step)const
	{
		triangle newT = *this;
		newT.a1.z += step;
		newT.a2.z += step;
		newT.a3.z += step;
		return newT;
	}

	inline __host__ __device__ Vec normal ()const
	{
		return _normal;
	}

	bool intersectSpher (const Ray &r) const
	{ // returns distance, 0 if nohit 
		//Vec op = o - r.o; // Solve t^2*d.d + 2*t*(o-p).d + (o-p).(o-p)-R^2 = 0 
		//double t,  b = op.dot (r.d), det = b*b - op.dot (op) + rad*rad;
		//if (det + EPSILON <0) return false; else det = sqrt (det);
		//return (t = b - det)>EPSILON ? t : ((t = b + det) > EPSILON ? t : 0);
		//
		
		
		//a == 1; // because rdir must be normalized
		Vec k = r.o - o;
		double b = k.dot ( r.d);
		double c =k. dot ( k) - rad*rad;
		double d = b*b - c;

		if (d-EPSILON >= 0)
		{
			double sqrtfd = sqrt (d);
			// t, a == 1
			double t1 = -b + sqrtfd;
			double t2 = -b - sqrtfd;
			
		    double min_t = t1<t2 ? t1 : t2;//min (t1, t2);
			double max_t = t1>t2 ? t1 : t2; //max (t1, t2);
			
			double t = (min_t >= 0) ? min_t : max_t;
			//tResult = t;
			return (t > 0);
		}
		else
		{
			return true;
		}
		//
	}

#ifdef  standartTest
	inline __host__ __device__ bool		intersect (const Ray &r, Vec & hit) const
	{
		double k = 0;								//plane coefficient

		Vec pl = this->normal ().normalization();	//normal of flat for this object
		double D = -pl.dot (a1);					//point's projection on normal 
		double nu = pl.dot (r.o) + D;				//numerator 
		double den = pl.dot (r.d);					//denominator
		if (abs (den) - EPSILON<0) return false;
		k = -(nu) / (den);							
		hit = r.d*k + r.o;							//hit point
		//verifications
		Vec hitor = hit - r.o;						//verification that the
		double heor = hitor.dot (r.d) - EPSILON;	//point along the ray path

		bool t = PointInTriangle (hit, a1, a2, a3); //verification that the
		//////////////////////////////////////////////point in triangle

		if (t && heor > 0)return true;				//verification access
		else			return false;

	}
	inline __host__ __device__ bool		SameSide (const Vec p1, const Vec p2, const Vec a, const Vec b)const
	{
		Vec cp1 = (b - a).cross (p1 - a);
		Vec cp2 = (b - a).cross (p2 - a);
		double d = cp1.dot (cp2);
		if (d + EPSILON >= 0) return true;
		else return false;
	}
	inline __host__ __device__ bool		PointInTriangle (const Vec p, const Vec a, const Vec b, const Vec c)const
	{
		if (SameSide (p, a, b, c) && 
			SameSide (p, b, a, c) && 
			SameSide (p, c, a, b))  
			 return true;
		else return false;
	}
#endif
	
#ifdef  baricentTest

	/* code rewritten to do tests on the sign of the determinant */
	/* the division is before the test of the sign of the det    */
	/* and one CROSS has been moved out from the if-else if-else */
	inline __host__ __device__ bool intersect (const Ray &r, Vec & hit) const
	{
		Vec edge1, edge2, tvec, pvec, qvec;
		double det, inv_det;
		double u, v, t;
		/* find vectors for two edges sharing vert0 */
		edge1 = a2 - a1;
		edge2 = a3 - a1;

		/* begin calculating determinant - also used to calculate U parameter */
		pvec = r.d%edge2;

		/* if determinant is near zero, ray lies in plane of triangle */
		det = edge1.dot (pvec);

		/* calculate distance from vert0 to ray origin */
		tvec = r.o - a1;
		inv_det = 1.0 / det;

		qvec = tvec%edge1;

		if (det > EPSILON)
		{
			u = tvec.dot (pvec);
			if (u - EPSILON < 0.0 || u > det - EPSILON)
				return false;

			/* calculate V parameter and test bounds */
			v = r.d.dot (qvec);
			if (v - EPSILON < 0.0 || u + v > det - EPSILON)
				return false;

		}
		else if (det < -EPSILON)
		{
			/* calculate U parameter and test bounds */
			u = tvec.dot (pvec);
			if (u + EPSILON > 0.0 || u - EPSILON < det)
				return false;

			/* calculate V parameter and test bounds */
			v = r.d.dot (qvec);
			if (v + EPSILON > 0.0 || u + v - EPSILON < det)
				return false;
		}
		else return false;  /* ray is parallell to the plane of the triangle */

		t = edge2.dot (qvec)*inv_det;
		u = inv_det;
		v = inv_det;
		hit = r.o + r.d*t;
		return true;
	}
#endif
};
struct camera
{
	Ray cameraLocation;
	Vec cameraXangle;
	Vec cameraYangle;
	camera (Ray cameraLocation_, Vec cameraXangle_) :
		cameraLocation (cameraLocation_),
		cameraXangle (cameraXangle_)
	{
		cameraYangle = (cameraXangle_.cross (cameraLocation_.d)).normalization ()*.5135;
	}
};
struct imgSettings
{
	unsigned int w;//width
	unsigned int h;//hight
	unsigned int samps;//sampels per pixel
	imgSettings (unsigned int w_, unsigned int h_, unsigned int samps_) :w (w_), h (h_), samps (samps_){}
};
struct world
{
	triangle*objects;
	Vec	*lights;
	unsigned int objCount;
	unsigned int lightsCount;
	world ();
	world (const unsigned int objCount_, const unsigned int lightsCount_,
		const triangle*objects_, const Vec	*lights_) :
		objCount (objCount_),
		lightsCount (lightsCount_)
	{
		objects = (triangle *)malloc (objCount_*sizeof(triangle));
		lights = (Vec *)malloc (lightsCount_*sizeof(Vec));
		for (unsigned int i = 0; i < objCount; ++i)
			objects[i] = objects_[i];
		for (unsigned int i = 0; i < lightsCount; ++i)
			lights[i] = lights_[i];
	}

};

 //world CUDA_WORLD();
triangle * CUDA_objects;
//enum Refl_t { DIFF, SPEC, REFR };  // material types, used in distanceiance() 
#pragma pack (push)
#pragma pack(1)
struct bmpinfo
{

	unsigned char signature1 = 'B';//1 byte
	unsigned char signature2 = 'M';//1 byte
	unsigned int fileSize;//4 byte
	unsigned int reserved = 0;//4 byte
	unsigned int offset = 14 + 40;//4 byte
	unsigned int headerSize = 40;//4 byte
	unsigned int width;//4 byte
	unsigned int height;//4 byte
	unsigned short colorPlanes = 1;//2 byte
	unsigned short bpp = 24;//2 byte
	unsigned int compression = 0;//4 byte
	unsigned int imgSize;//4 byte
	unsigned int resolutionHor = 2795;//4 byte
	unsigned int resolutionVer = 2795;//4 byte
	unsigned int pltColors = 0;//4 byte
	unsigned int impColors = 0;//4 byte
	bmpinfo (unsigned int _width, unsigned int _height)
	{
		width = _width;
		height = _height;
		fileSize = 14 + 40 + height * width * 3;
		imgSize = height *  width * 4;

	}
};
#pragma pack(pop)

Vec lights[] = { Vec (7, 70, 25),
				Vec (60, 7, 7) };
triangle obj[] = {
	triangle (Vec (37, 25, -4), Vec (39, 25, -4), Vec (39, 27, -4), Vec (.0, .0, .50), 0),//back

	triangle (Vec (5, 5, 5), Vec (6, 5, 5), Vec (6, 6, 5), Vec (0.6, 0.6, 0.6), 0.91),
	triangle (Vec (6, 5, 5), Vec (7, 5, 5), Vec (7, 6, 5), Vec (0.6, 0.6, 0.6), 0.92),
	triangle (Vec (7, 5, 5), Vec (8, 5, 5), Vec (8, 6, 5), Vec (0.6, 0.6, 0.6), 0.93),
	triangle (Vec (8, 5, 5), Vec (9, 5, 5), Vec (9, 6, 5), Vec (0.6, 0.6, 0.6), 0.94),
	triangle (Vec (9, 5, 5), Vec (10, 5, 5), Vec (10, 6, 5), Vec (0.6, 0.6, 0.6), 0.95),
	triangle (Vec (10, 5, 5), Vec (11, 5, 5), Vec (11, 6, 5), Vec (0.6, 0.6, 0.6), 0.96),
	triangle (Vec (11, 5, 5), Vec (12, 5, 5), Vec (12, 6, 5), Vec (0.6, 0.6, 0.6), 0.97),
	triangle (Vec (12, 5, 5), Vec (13, 5, 5), Vec (13, 6, 5), Vec (0.6, 0.6, 0.6), 0.98),
	triangle (Vec (13, 5, 5), Vec (14, 5, 5), Vec (14, 6, 5), Vec (0.6, 0.6, 0.6), 0.99),
	triangle (Vec (14, 5, 5), Vec (15, 5, 5), Vec (15, 6, 5), Vec (0.6, 0.6, 0.6), 0.90),

	triangle (Vec (6, 5, 5), Vec (6, 6, 5), Vec (7, 6, 5), Vec (0.6, 0.6, 0.6), 0.89),
	triangle (Vec (7, 5, 5), Vec (7, 6, 5), Vec (8, 6, 5), Vec (0.6, 0.6, 0.6), 0.88),
	triangle (Vec (8, 5, 5), Vec (8, 6, 5), Vec (9, 6, 5), Vec (0.6, 0.6, 0.6), 0.87),
	triangle (Vec (9, 5, 5), Vec (9, 6, 5), Vec (10, 6, 5), Vec (0.6, 0.6, 0.6), 0.86),
	triangle (Vec (10, 5, 5), Vec (10, 6, 5), Vec (11, 6, 5), Vec (0.6, 0.6, 0.6), 0.85),
	triangle (Vec (11, 5, 5), Vec (11, 6, 5), Vec (12, 6, 5), Vec (0.6, 0.6, 0.6), 0.84),
	triangle (Vec (12, 5, 5), Vec (12, 6, 5), Vec (13, 6, 5), Vec (0.6, 0.6, 0.6), 0.83),
	triangle (Vec (13, 5, 5), Vec (13, 6, 5), Vec (14, 6, 5), Vec (0.6, 0.6, 0.6), 0.82),
	triangle (Vec (14, 5, 5), Vec (14, 6, 5), Vec (15, 6, 5), Vec (0.6, 0.6, 0.6), 0.81),

	triangle (Vec (6, 6, 5), Vec (7, 6, 5), Vec (7, 7, 5), Vec (0.6, 0.6, 0.6), 0.89),
	triangle (Vec (7, 6, 5), Vec (8, 6, 5), Vec (8, 7, 5), Vec (0.6, 0.6, 0.6), 0.88),
	triangle (Vec (8, 6, 5), Vec (9, 6, 5), Vec (9, 7, 5), Vec (0.6, 0.6, 0.6), 0.87),
	triangle (Vec (9, 6, 5), Vec (10, 6, 5), Vec (10, 7, 5), Vec (0.6, 0.6, 0.6), 0.86),
	triangle (Vec (10, 6, 5), Vec (11, 6, 5), Vec (11, 7, 5), Vec (0.6, 0.6, 0.6), 0.85),
	triangle (Vec (11, 6, 5), Vec (12, 6, 5), Vec (12, 7, 5), Vec (0.6, 0.6, 0.6), 0.84),
	triangle (Vec (12, 6, 5), Vec (13, 6, 5), Vec (13, 7, 5), Vec (0.6, 0.6, 0.6), 0.83),
	triangle (Vec (13, 6, 5), Vec (14, 6, 5), Vec (14, 7, 5), Vec (0.6, 0.6, 0.6), 0.82),
	triangle (Vec (14, 6, 5), Vec (15, 6, 5), Vec (15, 7, 5), Vec (0.6, 0.6, 0.6), 0.81),

	triangle (Vec (7, 6, 5), Vec (7, 7, 5), Vec (8, 7, 5), Vec (0.6, 0.6, 0.6), 0.79),
	triangle (Vec (8, 6, 5), Vec (8, 7, 5), Vec (9, 7, 5), Vec (0.6, 0.6, 0.6), 0.78),
	triangle (Vec (9, 6, 5), Vec (9, 7, 5), Vec (10, 7, 5), Vec (0.6, 0.6, 0.6), 0.76),
	triangle (Vec (10, 6, 5), Vec (10, 7, 5), Vec (11, 7, 5), Vec (0.6, 0.6, 0.6), 0.75),
	triangle (Vec (11, 6, 5), Vec (11, 7, 5), Vec (12, 7, 5), Vec (0.6, 0.6, 0.6), 0.74),
	triangle (Vec (12, 6, 5), Vec (12, 7, 5), Vec (13, 7, 5), Vec (0.6, 0.6, 0.6), 0.73),
	triangle (Vec (13, 6, 5), Vec (13, 7, 5), Vec (14, 7, 5), Vec (0.6, 0.6, 0.6), 0.72),
	triangle (Vec (14, 6, 5), Vec (14, 7, 5), Vec (15, 7, 5), Vec (0.6, 0.6, 0.6), 0.71),

	triangle (Vec (7, 7, 5), Vec (8, 7, 5), Vec (8, 8, 5), Vec (0.6, 0.6, 0.6), 0.79),
	triangle (Vec (8, 7, 5), Vec (9, 7, 5), Vec (9, 8, 5), Vec (0.6, 0.6, 0.6), 0.78),
	triangle (Vec (9, 7, 5), Vec (10, 7, 5), Vec (10, 8, 5), Vec (0.6, 0.6, 0.6), 0.76),
	triangle (Vec (10, 7, 5), Vec (11, 7, 5), Vec (11, 8, 5), Vec (0.6, 0.6, 0.6), 0.75),
	triangle (Vec (11, 7, 5), Vec (12, 7, 5), Vec (12, 8, 5), Vec (0.6, 0.6, 0.6), 0.74),
	triangle (Vec (12, 7, 5), Vec (13, 7, 5), Vec (13, 8, 5), Vec (0.6, 0.6, 0.6), 0.73),
	triangle (Vec (13, 7, 5), Vec (14, 7, 5), Vec (14, 8, 5), Vec (0.6, 0.6, 0.6), 0.72),
	triangle (Vec (14, 7, 5), Vec (15, 7, 5), Vec (15, 8, 5), Vec (0.6, 0.6, 0.6), 0.71),

	triangle (Vec (8, 7, 5), Vec (8, 8, 5), Vec (9, 8, 5), Vec (0.6, 0.6, 0.6), 0.69),
	triangle (Vec (9, 7, 5), Vec (9, 8, 5), Vec (10, 8, 5), Vec (0.6, 0.6, 0.6), 0.68),
	triangle (Vec (10, 7, 5), Vec (10, 8, 5), Vec (11, 8, 5), Vec (0.6, 0.6, 0.6), 0.67),
	triangle (Vec (11, 7, 5), Vec (11, 8, 5), Vec (12, 8, 5), Vec (0.6, 0.6, 0.6), 0.66),
	triangle (Vec (12, 7, 5), Vec (12, 8, 5), Vec (13, 8, 5), Vec (0.6, 0.6, 0.6), 0.65),
	triangle (Vec (13, 7, 5), Vec (13, 8, 5), Vec (14, 8, 5), Vec (0.6, 0.6, 0.6), 0.64),
	triangle (Vec (14, 7, 5), Vec (14, 8, 5), Vec (15, 8, 5), Vec (0.6, 0.6, 0.6), 0.63),

	triangle (Vec (8, 8, 5), Vec (9, 8, 5), Vec (9, 9, 5), Vec (0.6, 0.6, 0.6), 0.69),
	triangle (Vec (9, 8, 5), Vec (10, 8, 5), Vec (10, 9, 5), Vec (0.6, 0.6, 0.6), 0.68),
	triangle (Vec (10, 8, 5), Vec (11, 8, 5), Vec (11, 9, 5), Vec (0.6, 0.6, 0.6), 0.67),
	triangle (Vec (11, 8, 5), Vec (12, 8, 5), Vec (12, 9, 5), Vec (0.6, 0.6, 0.6), 0.66),
	triangle (Vec (12, 8, 5), Vec (13, 8, 5), Vec (13, 9, 5), Vec (0.6, 0.6, 0.6), 0.65),
	triangle (Vec (13, 8, 5), Vec (14, 8, 5), Vec (14, 9, 5), Vec (0.6, 0.6, 0.6), 0.64),
	triangle (Vec (14, 8, 5), Vec (15, 8, 5), Vec (15, 9, 5), Vec (0.6, 0.6, 0.6), 0.63),

	triangle (Vec (9, 8, 5), Vec (9, 9, 5), Vec (10, 9, 5), Vec (0.6, 0.6, 0.6), 0.58),
	triangle (Vec (10, 8, 5), Vec (10, 9, 5), Vec (11, 9, 5), Vec (0.6, 0.6, 0.6), 0.57),
	triangle (Vec (11, 8, 5), Vec (11, 9, 5), Vec (12, 9, 5), Vec (0.6, 0.6, 0.6), 0.56),
	triangle (Vec (12, 8, 5), Vec (12, 9, 5), Vec (13, 9, 5), Vec (0.6, 0.6, 0.6), 0.55),
	triangle (Vec (13, 8, 5), Vec (13, 9, 5), Vec (14, 9, 5), Vec (0.6, 0.6, 0.6), 0.54),
	triangle (Vec (14, 8, 5), Vec (14, 9, 5), Vec (15, 9, 5), Vec (0.6, 0.6, 0.6), 0.53),

	triangle (Vec (9, 9, 5), Vec (10, 9, 5), Vec (10, 10, 5), Vec (0.6, 0.6, 0.6), 0.58),
	triangle (Vec (10, 9, 5), Vec (11, 9, 5), Vec (11, 10, 5), Vec (0.6, 0.6, 0.6), 0.57),
	triangle (Vec (11, 9, 5), Vec (12, 9, 5), Vec (12, 10, 5), Vec (0.6, 0.6, 0.6), 0.56),
	triangle (Vec (12, 9, 5), Vec (13, 9, 5), Vec (13, 10, 5), Vec (0.6, 0.6, 0.6), 0.55),
	triangle (Vec (13, 9, 5), Vec (14, 9, 5), Vec (14, 10, 5), Vec (0.6, 0.6, 0.6), 0.54),
	triangle (Vec (14, 9, 5), Vec (15, 9, 5), Vec (15, 10, 5), Vec (0.6, 0.6, 0.6), 0.53),

	triangle (Vec (10, 9, 5), Vec (10, 10, 5), Vec (11, 10, 5), Vec (0.6, 0.6, 0.6), 0.46),
	triangle (Vec (11, 9, 5), Vec (11, 10, 5), Vec (12, 10, 5), Vec (0.6, 0.6, 0.6), 0.47),
	triangle (Vec (12, 9, 5), Vec (12, 10, 5), Vec (13, 10, 5), Vec (0.6, 0.6, 0.6), 0.48),
	triangle (Vec (13, 9, 5), Vec (13, 10, 5), Vec (14, 10, 5), Vec (0.6, 0.6, 0.6), 0.49),
	triangle (Vec (14, 9, 5), Vec (14, 10, 5), Vec (15, 10, 5), Vec (0.6, 0.6, 0.6), 0.40),

	triangle (Vec (10, 10, 5), Vec (11, 10, 5), Vec (11, 11, 5), Vec (0.6, 0.6, 0.6), 0.46),
	triangle (Vec (11, 10, 5), Vec (12, 10, 5), Vec (12, 11, 5), Vec (0.6, 0.6, 0.6), 0.47),
	triangle (Vec (12, 10, 5), Vec (13, 10, 5), Vec (13, 11, 5), Vec (0.6, 0.6, 0.6), 0.48),
	triangle (Vec (13, 10, 5), Vec (14, 10, 5), Vec (14, 11, 5), Vec (0.6, 0.6, 0.6), 0.49),
	triangle (Vec (14, 10, 5), Vec (15, 10, 5), Vec (15, 11, 5), Vec (0.6, 0.6, 0.6), 0.41),

	triangle (Vec (11, 10, 5), Vec (11, 11, 5), Vec (12, 11, 5), Vec (0.6, 0.6, 0.6), 0.37),
	triangle (Vec (12, 10, 5), Vec (12, 11, 5), Vec (13, 11, 5), Vec (0.6, 0.6, 0.6), 0.38),
	triangle (Vec (13, 10, 5), Vec (13, 11, 5), Vec (14, 11, 5), Vec (0.6, 0.6, 0.6), 0.39),
	triangle (Vec (14, 10, 5), Vec (14, 11, 5), Vec (15, 11, 5), Vec (0.6, 0.6, 0.6), 0.30),

	triangle (Vec (11, 11, 5), Vec (12, 11, 5), Vec (12, 12, 5), Vec (0.6, 0.6, 0.6), 0.37),
	triangle (Vec (12, 11, 5), Vec (13, 11, 5), Vec (13, 12, 5), Vec (0.6, 0.6, 0.6), 0.38),
	triangle (Vec (13, 11, 5), Vec (14, 11, 5), Vec (14, 12, 5), Vec (0.6, 0.6, 0.6), 0.39),
	triangle (Vec (14, 11, 5), Vec (15, 11, 5), Vec (15, 12, 5), Vec (0.6, 0.6, 0.6), 0.31),

	triangle (Vec (12, 11, 5), Vec (12, 12, 5), Vec (13, 12, 5), Vec (0.6, 0.6, 0.6), 0.28),
	triangle (Vec (13, 11, 5), Vec (13, 12, 5), Vec (14, 12, 5), Vec (0.6, 0.6, 0.6), 0.29),
	triangle (Vec (14, 11, 5), Vec (14, 12, 5), Vec (15, 12, 5), Vec (0.6, 0.6, 0.6), 0.20),

	triangle (Vec (12, 12, 5), Vec (13, 12, 5), Vec (13, 13, 5), Vec (0.6, 0.6, 0.6), 0.28),
	triangle (Vec (13, 12, 5), Vec (14, 12, 5), Vec (14, 13, 5), Vec (0.6, 0.6, 0.6), 0.29),
	triangle (Vec (14, 12, 5), Vec (15, 12, 5), Vec (15, 13, 5), Vec (0.6, 0.6, 0.6), 0.21),

	triangle (Vec (13, 12, 5), Vec (13, 13, 5), Vec (14, 13, 5), Vec (0.6, 0.6, 0.6), 0.19),
	triangle (Vec (14, 12, 5), Vec (14, 13, 5), Vec (15, 13, 5), Vec (0.6, 0.6, 0.6), 0.10),

	triangle (Vec (14, 13, 5), Vec (14, 14, 5), Vec (15, 14, 5), Vec (0.6, 0.6, 0.6), 0.05),

	triangle (Vec (13, 13, 5), Vec (14, 13, 5), Vec (14, 14, 5), Vec (0.6, 0.6, 0.6), 0.19),
	triangle (Vec (14, 13, 5), Vec (15, 13, 5), Vec (15, 14, 5), Vec (0.6, 0.6, 0.6), 0.11),

	triangle (Vec (14, 14, 5), Vec (15, 14, 5), Vec (15, 15, 5), Vec (0.6, 0.6, 0.6), 0.04),



	triangle (Vec (5, 5, 5), Vec (15, 5, 5), Vec (15, 5, 15), Vec (.50, .0, .0), 0),//down
	triangle (Vec (5, 5, 5), Vec (5, 5, 15), Vec (15, 5, 15), Vec (.50, .0, .0), 0),//down
	//triangle(Vec(5,15,5	),Vec(15,15,5	),Vec(15,15,15	),Vec(.50,.15,.0),0),//up
	//triangle(Vec(5,15,5	),Vec(5,15,15	),Vec(15,15,15	),Vec(.50,.0,.15),0),//up																	 
	//triangle (Vec (5, 5, 5), Vec (15, 5, 5), Vec (15, 15, 5), Vec (.10, .10, .10), 0.85),//back
	triangle (Vec (5, 5, 5), Vec (5, 15, 5), Vec (15, 15, 5), Vec (.10, .10, .10), 0.85),//back
	//triangle(Vec(5,5,15	),Vec(15,5,15	),Vec(15,15,15	),Vec(.0,.10,.50),0),//front
	//triangle(Vec(5,5,15	),Vec(5,15,15	),Vec(15,15,15	),Vec(.10,.0,.50),0),//front																	 
	triangle (Vec (5, 5, 5), Vec (5, 5, 15), Vec (5, 15, 15), Vec (.50, .50, .0), 0.6),//left
	triangle (Vec (5, 5, 5), Vec (5, 15, 5), Vec (5, 15, 15), Vec (.0, .50, .50), 0),//left
	//triangle(Vec(15,5,5	),Vec(15,5,15	),Vec(15,15,15	),Vec(.10,.50,.0),0.60),//right
	//triangle(Vec(15,5,5	),Vec(15,15,5	),Vec(15,15,15	),Vec(.0,.50,.10),0.60),//right

	triangle (Vec (5, 5, 5), Vec (15, 5, 5), Vec (15, 5, 15), Vec (.50, .0, .0), 0).moveX (15),//down
	triangle (Vec (5, 5, 5), Vec (5, 5, 15), Vec (15, 5, 15), Vec (.50, .0, .0), 0).moveX (15),//down
	triangle (Vec (5, 15, 5), Vec (15, 15, 5), Vec (15, 15, 15), Vec (.50, .15, .0), 0).moveX (15),//up
	triangle (Vec (5, 15, 5), Vec (5, 15, 15), Vec (15, 15, 15), Vec (.50, .0, .15), 0).moveX (15),//up																	 
	triangle (Vec (5, 5, 5), Vec (15, 5, 5), Vec (15, 15, 5), Vec (.0, .0, .50), 0).moveX (15),//back
	triangle (Vec (5, 5, 5), Vec (5, 15, 5), Vec (15, 15, 5), Vec (.0, .0, .50), 0).moveX (15),//back
	triangle (Vec (5, 5, 15), Vec (15, 5, 15), Vec (15, 15, 15), Vec (.0, .10, .50), 0).moveX (15),//front
	triangle (Vec (5, 5, 15), Vec (5, 15, 15), Vec (15, 15, 15), Vec (.10, .0, .50), 0).moveX (15),//front																	 
	triangle (Vec (5, 5, 5), Vec (5, 5, 15), Vec (5, 15, 15), Vec (.0, .50, .0), 0).moveX (15),//left
	triangle (Vec (5, 5, 5), Vec (5, 15, 5), Vec (5, 15, 15), Vec (.0, .50, .0), 0).moveX (15),//left
	triangle (Vec (15, 5, 5), Vec (15, 5, 15), Vec (15, 15, 15), Vec (.10, .50, .0), 0).moveX (15),//right
	triangle (Vec (15, 5, 5), Vec (15, 15, 5), Vec (15, 15, 15), Vec (.0, .50, .10), 0).moveX (15),//right

	triangle (Vec (5, 5, 5), Vec (15, 5, 5), Vec (15, 5, 15), Vec (.50, .0, .0), 0).moveZ (-15),//down
	triangle (Vec (5, 5, 5), Vec (5, 5, 15), Vec (15, 5, 15), Vec (.50, .0, .0), 0).moveZ (-15),//down
	triangle (Vec (5, 15, 5), Vec (15, 15, 5), Vec (15, 15, 15), Vec (.50, .15, .0), 0).moveZ (-15),//up
	triangle (Vec (5, 15, 5), Vec (5, 15, 15), Vec (15, 15, 15), Vec (.50, .0, .15), 0).moveZ (-15),//up																	 	   
	triangle (Vec (5, 5, 5), Vec (15, 5, 5), Vec (15, 15, 5), Vec (.0, .0, .50), 0).moveZ (-15),//back
	triangle (Vec (5, 5, 5), Vec (5, 15, 5), Vec (15, 15, 5), Vec (.0, .0, .50), 0).moveZ (-15),//back
	triangle (Vec (5, 5, 15), Vec (15, 5, 15), Vec (15, 15, 15), Vec (.0, .10, .50), 0).moveZ (-15),//front
	triangle (Vec (5, 5, 15), Vec (5, 15, 15), Vec (15, 15, 15), Vec (.10, .0, .50), 0).moveZ (-15),//front																	 	  
	triangle (Vec (5, 5, 5), Vec (5, 5, 15), Vec (5, 15, 15), Vec (.0, .50, .0), 0).moveZ (-15),//left
	triangle (Vec (5, 5, 5), Vec (5, 15, 5), Vec (5, 15, 15), Vec (.0, .50, .0), 0).moveZ (-15),//left
	triangle (Vec (15, 5, 5), Vec (15, 5, 15), Vec (15, 15, 15), Vec (.10, .50, .0), 0).moveZ (-15),//right
	triangle (Vec (15, 5, 5), Vec (15, 15, 5), Vec (15, 15, 15), Vec (.0, .50, .10), 0).moveZ (-15),//right

	triangle (Vec (5, 5, 5), Vec (15, 5, 5), Vec (15, 5, 15), Vec (.50, .0, .0), 0).moveX (15).moveZ (-15),//down
	triangle (Vec (5, 5, 5), Vec (5, 5, 15), Vec (15, 5, 15), Vec (.50, .0, .0), 0).moveX (15).moveZ (-15),//down
	triangle (Vec (5, 15, 5), Vec (15, 15, 5), Vec (15, 15, 15), Vec (.50, .15, .0), 0).moveX (15).moveZ (-15),//up
	triangle (Vec (5, 15, 5), Vec (5, 15, 15), Vec (15, 15, 15), Vec (.50, .0, .15), 0).moveX (15).moveZ (-15),//up																	 
	triangle (Vec (5, 5, 5), Vec (15, 5, 5), Vec (15, 15, 5), Vec (.0, .0, .50), 0).moveX (15).moveZ (-15),//back
	triangle (Vec (5, 5, 5), Vec (5, 15, 5), Vec (15, 15, 5), Vec (.0, .0, .50), 0).moveX (15).moveZ (-15),//back
	triangle (Vec (5, 5, 15), Vec (15, 5, 15), Vec (15, 15, 15), Vec (.0, .10, .50), 0).moveX (15).moveZ (-15),//front
	triangle (Vec (5, 5, 15), Vec (5, 15, 15), Vec (15, 15, 15), Vec (.10, .0, .50), 0).moveX (15).moveZ (-15),//front																	 
	triangle (Vec (5, 5, 5), Vec (5, 5, 15), Vec (5, 15, 15), Vec (.0, .50, .0), 0).moveX (15).moveZ (-15),//left
	triangle (Vec (5, 5, 5), Vec (5, 15, 5), Vec (5, 15, 15), Vec (.0, .50, .0), 0).moveX (15).moveZ (-15),//left
	triangle (Vec (15, 5, 5), Vec (15, 5, 15), Vec (15, 15, 15), Vec (.10, .50, .0), 0).moveX (15).moveZ (-15),//right
	triangle (Vec (15, 5, 5), Vec (15, 15, 5), Vec (15, 15, 15), Vec (.0, .50, .10), 0).moveX (15).moveZ (-15),//right

	triangle (Vec (5, 5, 5), Vec (15, 5, 5), Vec (15, 5, 15), Vec (.50, .0, .0), 0).moveY (15),//down
	triangle (Vec (5, 5, 5), Vec (5, 5, 15), Vec (15, 5, 15), Vec (.50, .0, .0), 0).moveY (15),//down
	triangle (Vec (5, 15, 5), Vec (15, 15, 5), Vec (15, 15, 15), Vec (.50, .15, .0), 0).moveY (15),//up
	triangle (Vec (5, 15, 5), Vec (5, 15, 15), Vec (15, 15, 15), Vec (.50, .0, .15), 0).moveY (15),//up																	 
	triangle (Vec (5, 5, 5), Vec (15, 5, 5), Vec (15, 15, 5), Vec (.0, .0, .50), 0).moveY (15),//back
	triangle (Vec (5, 5, 5), Vec (5, 15, 5), Vec (15, 15, 5), Vec (.0, .0, .50), 0).moveY (15),//back
	triangle (Vec (5, 5, 15), Vec (15, 5, 15), Vec (15, 15, 15), Vec (.0, .10, .50), 0).moveY (15),//front
	triangle (Vec (5, 5, 15), Vec (5, 15, 15), Vec (15, 15, 15), Vec (.10, .0, .50), 0).moveY (15),//front																	 
	triangle (Vec (5, 5, 5), Vec (5, 5, 15), Vec (5, 15, 15), Vec (.0, .50, .0), 0).moveY (15),//left
	triangle (Vec (5, 5, 5), Vec (5, 15, 5), Vec (5, 15, 15), Vec (.0, .50, .0), 0).moveY (15),//left
	triangle (Vec (15, 5, 5), Vec (15, 5, 15), Vec (15, 15, 15), Vec (.10, .50, .0), 0).moveY (15),//right
	triangle (Vec (15, 5, 5), Vec (15, 15, 5), Vec (15, 15, 15), Vec (.0, .50, .10), 0).moveY (15),//right

	triangle (Vec (5, 5, 5), Vec (15, 5, 5), Vec (15, 5, 15), Vec (.50, .0, .0), 0).moveX (15).moveY (15),//down
	triangle (Vec (5, 5, 5), Vec (5, 5, 15), Vec (15, 5, 15), Vec (.50, .0, .0), 0).moveX (15).moveY (15),//down
	triangle (Vec (5, 15, 5), Vec (15, 15, 5), Vec (15, 15, 15), Vec (.50, .15, .0), 0).moveX (15).moveY (15),//up
	triangle (Vec (5, 15, 5), Vec (5, 15, 15), Vec (15, 15, 15), Vec (.50, .0, .15), 0).moveX (15).moveY (15),//up																	 
	triangle (Vec (5, 5, 5), Vec (15, 5, 5), Vec (15, 15, 5), Vec (.0, .0, .50), 0).moveX (15).moveY (15),//back
	triangle (Vec (5, 5, 5), Vec (5, 15, 5), Vec (15, 15, 5), Vec (.0, .0, .50), 0).moveX (15).moveY (15),//back
	triangle (Vec (5, 5, 15), Vec (15, 5, 15), Vec (15, 15, 15), Vec (.0, .10, .50), 0).moveX (15).moveY (15),//front
	triangle (Vec (5, 5, 15), Vec (5, 15, 15), Vec (15, 15, 15), Vec (.10, .0, .50), 0).moveX (15).moveY (15),//front																	 
	triangle (Vec (5, 5, 5), Vec (5, 5, 15), Vec (5, 15, 15), Vec (.0, .50, .0), 0).moveX (15).moveY (15),//left
	triangle (Vec (5, 5, 5), Vec (5, 15, 5), Vec (5, 15, 15), Vec (.0, .50, .0), 0).moveX (15).moveY (15),//left
	triangle (Vec (15, 5, 5), Vec (15, 5, 15), Vec (15, 15, 15), Vec (.10, .50, .0), 0).moveX (15).moveY (15),//right
	triangle (Vec (15, 5, 5), Vec (15, 15, 5), Vec (15, 15, 15), Vec (.0, .50, .10), 0).moveX (15).moveY (15),//right

	triangle (Vec (5, 5, 5), Vec (15, 5, 5), Vec (15, 5, 15), Vec (.50, .0, .0), 0).moveZ (-15).moveY (15),//down
	triangle (Vec (5, 5, 5), Vec (5, 5, 15), Vec (15, 5, 15), Vec (.50, .0, .0), 0).moveZ (-15).moveY (15),//down
	triangle (Vec (5, 15, 5), Vec (15, 15, 5), Vec (15, 15, 15), Vec (.50, .15, .0), 0).moveZ (-15).moveY (15),//up
	triangle (Vec (5, 15, 5), Vec (5, 15, 15), Vec (15, 15, 15), Vec (.50, .0, .15), 0).moveZ (-15).moveY (15),//up																	 	   
	triangle (Vec (5, 5, 5), Vec (15, 5, 5), Vec (15, 15, 5), Vec (.0, .0, .50), 0).moveZ (-15).moveY (15),//back
	triangle (Vec (5, 5, 5), Vec (5, 15, 5), Vec (15, 15, 5), Vec (.0, .0, .50), 0).moveZ (-15).moveY (15),//back
	triangle (Vec (5, 5, 15), Vec (15, 5, 15), Vec (15, 15, 15), Vec (.0, .10, .50), 0).moveZ (-15).moveY (15),//front
	triangle (Vec (5, 5, 15), Vec (5, 15, 15), Vec (15, 15, 15), Vec (.10, .0, .50), 0).moveZ (-15).moveY (15),//front																	 	  
	triangle (Vec (5, 5, 5), Vec (5, 5, 15), Vec (5, 15, 15), Vec (.0, .50, .0), 0).moveZ (-15).moveY (15),//left
	triangle (Vec (5, 5, 5), Vec (5, 15, 5), Vec (5, 15, 15), Vec (.0, .50, .0), 0).moveZ (-15).moveY (15),//left
	triangle (Vec (15, 5, 5), Vec (15, 5, 15), Vec (15, 15, 15), Vec (.10, .50, .0), 0).moveZ (-15).moveY (15),//right
	triangle (Vec (15, 5, 5), Vec (15, 15, 5), Vec (15, 15, 15), Vec (.0, .50, .10), 0).moveZ (-15).moveY (15),//right

	triangle (Vec (5, 5, 5), Vec (15, 5, 5), Vec (15, 5, 15), Vec (.50, .0, .0), 0).moveX (15).moveZ (-15).moveY (15),//down
	triangle (Vec (5, 5, 5), Vec (5, 5, 15), Vec (15, 5, 15), Vec (.50, .0, .0), 0).moveX (15).moveZ (-15).moveY (15),//down
	triangle (Vec (5, 15, 5), Vec (15, 15, 5), Vec (15, 15, 15), Vec (.50, .15, .0), 0).moveX (15).moveZ (-15).moveY (15),//up
	triangle (Vec (5, 15, 5), Vec (5, 15, 15), Vec (15, 15, 15), Vec (.50, .0, .15), 0).moveX (15).moveZ (-15).moveY (15),//up																	 
	triangle (Vec (5, 5, 5), Vec (15, 5, 5), Vec (15, 15, 5), Vec (.0, .0, .50), 0).moveX (15).moveZ (-15).moveY (15),//back
	triangle (Vec (5, 5, 5), Vec (5, 15, 5), Vec (15, 15, 5), Vec (.0, .0, .50), 0).moveX (15).moveZ (-15).moveY (15),//back
	triangle (Vec (5, 5, 15), Vec (15, 5, 15), Vec (15, 15, 15), Vec (.0, .10, .50), 0).moveX (15).moveZ (-15).moveY (15),//front
	triangle (Vec (5, 5, 15), Vec (5, 15, 15), Vec (15, 15, 15), Vec (.10, .0, .50), 0).moveX (15).moveZ (-15).moveY (15),//front																	 
	triangle (Vec (5, 5, 5), Vec (5, 5, 15), Vec (5, 15, 15), Vec (.0, .50, .0), 0).moveX (15).moveZ (-15).moveY (15),//left
	triangle (Vec (5, 5, 5), Vec (5, 15, 5), Vec (5, 15, 15), Vec (.0, .50, .0), 0).moveX (15).moveZ (-15).moveY (15),//left
	triangle (Vec (15, 5, 5), Vec (15, 5, 15), Vec (15, 15, 15), Vec (.10, .50, .0), 0).moveX (15).moveZ (-15).moveY (15),//right
	triangle (Vec (15, 5, 5), Vec (15, 15, 5), Vec (15, 15, 15), Vec (.0, .50, .10), 0).moveX (15).moveZ (-15).moveY (15)//right

};


//struct Sphere {
//	double distance;       // distanceius 
//	Vec p, e, c;      // position, emission, color 
//	Refl_t refl;      // reflection type (DIFFuse, SPECular, REFRactive) 
//	Sphere(double distance_, Vec p_, Vec e_, Vec c_, Refl_t refl_) :
//		distance(distance_), p(p_), e(e_), c(c_), refl(refl_) {}
//	double intersect(const Ray &r) const 
//	{ // returns distance, 0 if nohit 
//		Vec op = p - r.o; // Solve t^2*d.d + 2*t*(o-p).d + (o-p).(o-p)-R^2 = 0 
//		double t, eps = 1e-4, b = op.dot(r.d), det = b*b - op.dot(op) + distance*distance;
//		if (det<0) return 0; else det = sqrt(det);
//		return (t = b - det)>eps ? t : ((t = b + det) > eps ? t : 0);
//	}
//};
//
//Sphere spheres[] = {//Scene: distanceius, position, emission, color, material 
//	Sphere(1e5, Vec(1e5 + 1,40.8,81.6), Vec(),Vec(.75,.25,.25),DIFF),//Left 
//	Sphere(1e5, Vec(-1e5 + 99,40.8,81.6),Vec(),Vec(.25,.25,.75),DIFF),//Rght 
//	Sphere(1e5, Vec(50,40.8, 1e5),     Vec(),Vec(.75,.75,.75),DIFF),//Back 
//	Sphere(1e5, Vec(50,40.8,-1e5 + 170), Vec(),Vec(),           DIFF),//Frnt 
//	Sphere(1e5, Vec(50, 1e5, 81.6),    Vec(),Vec(.75,.75,.75),DIFF),//Botm 
//	Sphere(1e5, Vec(50,-1e5 + 81.6,81.6),Vec(),Vec(.75,.75,.75),DIFF),//Top 
//	Sphere(16.5,Vec(27,16.5,47),       Vec(),Vec(1,1,1)*.999, SPEC),//Mirr 
//	Sphere(16.5,Vec(73,16.5,78),       Vec(),Vec(1,1,1)*.999, REFR),//Glas 
//	Sphere(600, Vec(50,681.6 - .27,81.6),Vec(12,12,12),  Vec(), DIFF) //Lite 
//};

inline double	clamp  (double x) { return x < 0 ? 0 : x>1 ? 1 : x; }
inline int		toInt  (double x) { return int  (pow (clamp (x), 1 / 2.2) * 255 + .5); }
inline char		toChar (double x) { return char (pow (clamp (x), 1 / 2.2) * 255); }



bool helpOrient (triangle t, Vec v1, Vec v2)
{

	Vec normalization = t.normal ();
	double x1 = normalization.dot (v1 - t.a1);
	double x2 = normalization.dot (v2 - t.a1);
	return x1*x2 + EPSILON >= 0;
}
/*
Returns true if input ray intersect some object
*/


inline __host__ __device__ void intersect (const triangle * objects, const unsigned int objCount,
	const Ray &r, double &t, int &id, Vec & hit, bool * isIntersect)
{
	Vec lockalHit;
	double  d; 
	t = INF;
	for (unsigned int i = objCount; i--;)
	{
		//bool isSpherIntersection = objects[i].intersectSpher (r);
		//if (isSpherIntersection)
		{
			bool isIntersection = objects[i].intersect (r, lockalHit);
			if (isIntersection)
			{
				d = lockalHit.distance (r.o);
				if (d < t)
				{
					t = d;
					id = i;
					hit = lockalHit;
				}
			}
		}
	}
	*isIntersect =  t < INF;
}
inline __host__ __device__ bool intersectHelper (const triangle * objects, const unsigned int objCount,
	const Ray &r, double &t, int &id, Vec & hit)
{
	bool  isIntersect = 0;
	intersect (objects, objCount, r, t, id, hit, &isIntersect);
	return isIntersect;
}
/*
Returns true if the light source is visible
*/
inline __host__ __device__ bool Visible (const  world & wrld, const Vec & hit, const Vec & light, const int & id)
{
	double distToLight = hit.distance (light);
	Vec hit1;
	Vec sub = light - hit;
	Ray r (hit, sub);
	double distanse;
	int id1 = -1;
	bool isIntersection = intersectHelper (wrld.objects, wrld.objCount, r, distanse, id1, hit1);
	if (isIntersection)
	{
		if (id == id1)return true;
		return false;
	}
	else return true;

}
inline __host__ __device__ Vec Shade (const Vec & hit, const Vec & light)
{
	//double ka = 0.1; //ambient coefficient
	return Vec (0.3, 0.3, 0.3);
}
/*
Return new reflected ray
*/
inline __host__ __device__ Ray reflect (const Ray & r,const  triangle & obj,const Vec & hit)
{
	Vec normal = obj.normal ();
	Vec iV = hit - r.o;//inputVector
	normal = normal.normalization ();
	Vec rV = iV - ((normal*(iV.dot (normal))) * 2);//reflectVector
	Ray reflect (hit, rV.normalization ());//new reflect ray
	return reflect;
}
/*
Recursively trace the input ray with a light source and reflection
*/
__host__ __device__ Vec RayTrace (const  world  & wrld,const Ray & ray,unsigned int deep)
{
	Vec color (0, 0, 0);
	int id = 0;
	Vec hit;// ������ �������
	double distanse ;
	bool isIntersection = intersectHelper (wrld.objects, wrld.objCount, ray, distanse, id, hit);
	
	if (!isIntersection)
		return color;
	triangle tr = wrld.objects[id];
	color = tr.c;

	unsigned int lC = wrld.lightsCount;
	for (unsigned int i = 0; i < lC; ++i)
	{//�������� ������������
		bool isVisible = Visible (wrld, hit, wrld.lights[i], id);
		if (isVisible)
		{
			Vec light = wrld.lights[i] - hit;
			double distancei = wrld.lights[i].distance (hit);
			double cos = abs ((light.dot (tr.normal ().normalization ())) / (distancei));
			color = color + color*(1 / (distancei*distancei));
		}
		else
		{
			color = color - color*0.8;
		}
	}

	//	

	if (tr.reflect > 0 && deep > 0)//������ ���������
	{
		Ray reflRay = reflect (ray, tr, hit);
		color = color*(1.0 - tr.reflect) + RayTrace (wrld, reflRay, deep--)*tr.reflect;
	}
	return color;
}
//Vec distanceiance(const Ray &r, int depth, unsigned short *Xi) 
//{
//	
//	double t;                               // distance to intersection 
//	int id = 0;                               // id of intersected object 
//	if (!intersect(r, t, id)) return Vec(); // if miss, return black 
//	const Sphere &obj = spheres[id];        // the hit object 
//	Vec x = r.o + r.d*t, n = (x - obj.p).normalization(), nl = n.dot(r.d) < 0 ? n : n*-1, f = obj.c;
//	double p = f.x > f.y && f.x>f.z ? f.x : f.y > f.z ? f.y : f.z; // max refl 
//	if (++depth > 5) 
//		if (erand48(Xi) < p) f = f*(1 / p); 
//	else return obj.e; //R.R. 
//	if (obj.refl == DIFF) 
//	{                  // Ideal DIFFUSE reflection 
//		double r1 = 2 * 3.14159265358979323846 *erand48(Xi), r2 = erand48(Xi), r2s = sqrt(r2);
//		Vec w = nl, u = ((fabs(w.x) > .1 ? Vec(0, 1) : Vec(1)) % w).normalization(), v = w%u;
//		Vec d = (u*cos(r1)*r2s + v*sin(r1)*r2s + w*sqrt(1 - r2)).normalization();
//		return obj.e + f.mult(distanceiance(Ray(x, d), depth, Xi));
//	}
//	else if (obj.refl == SPEC)            // Ideal SPECULAR reflection 
//		return obj.e + f.mult(distanceiance(Ray(x, r.d - n * 2 * n.dot(r.d)), depth, Xi));
//	Ray reflRay(x, r.d - n * 2 * n.dot(r.d));     // Ideal dielectric REFRACTION 
//	bool into = n.dot(nl) > 0;                // Ray from outside going in? 
//	double nc = 1, nt = 1.5, nnt = into ? nc / nt : nt / nc, ddn = r.d.dot(nl), cos2t;
//	if ((cos2t = 1 - nnt*nnt*(1 - ddn*ddn)) < 0)    // Total internal reflection 
//		return obj.e + f.mult(distanceiance(reflRay, depth, Xi));
//	Vec tdir = (r.d*nnt - n*((into ? 1 : -1)*(ddn*nnt + sqrt(cos2t)))).normalization();
//	double a = nt - nc, b = nt + nc, R0 = a*a / (b*b), c = 1 - (into ? -ddn : tdir.dot(n));
//	double Re = R0 + (1 - R0)*c*c*c*c*c, Tr = 1 - Re, P = .25 + .5*Re, RP = Re / P, TP = Tr / (1 - P);
//	return obj.e + f.mult(depth > 2 ? (erand48(Xi) < P ?   // Russian roulette 
//		distanceiance(reflRay, depth, Xi)*RP : distanceiance(Ray(x, tdir), depth, Xi)*TP) :
//		distanceiance(reflRay, depth, Xi)*Re + distanceiance(Ray(x, tdir), depth, Xi)*Tr);
//}
inline void fileOutput (Vec *c, const unsigned  int w, const unsigned int h)
{
#ifdef ppm
	FILE *f = fopen ("image.ppm", "w");         // Write image to PPM file. 
	fprintf (f, "P3\n%d %d\n%d\n", w, h, 255);
	for (unsigned int i = 0; i < w*h; ++i)
		fprintf (f, "%d %d %d ", toInt (c[i].x), toInt (c[i].y), toInt (c[i].z));
#endif

#ifdef bmp
	bmpinfo first (w, h);
	FILE *f = fopen ("temp1.bmp", "wb");  // 

	fwrite (&first, sizeof(bmpinfo), 1, f);
	int step = (w * 3 + 3) / 4 * 4;
	unsigned char *data = new unsigned char[step*h];

	for (int i = h - 1; i >= 0; i--)
	{
		unsigned char *p = data + step * (h - i);

		for (int j = 0; j < w; j++)
		{
			*p++ = toChar (c[i*w + j].z);
			*p++ = toChar (c[i*w + j].y);
			*p++ = toChar (c[i*w + j].x);
		}
	}

	fwrite (data, step*h, 1, f);
	delete[] data;


	//int n = sizeof(first);
	//char * b = new char[n];
	//memcpy (b, &first, n);
	//for (int i = 0; i < n; i++)fprintf (f, "%c", b[i]);
	////unsigned char x = 0;

	//for (int i = h - 1; i >= 0; i--)
	//{
	//	for (int j = 0; j < w; j++)
	//	{

	//		fprintf (f, "%c%c%c", toChar (c[i*w + j].x), toChar (c[i*w + j].z), toChar (c[i*w + j].y));//rbg
	//	}
	//}

	fclose (f);
#endif
}

Ray * RenderRayHelper (const camera & cam, const imgSettings & img)
{
	Vec r;
	int kFilter = 4;
#ifdef sampleInCenter
	kFilter = 1;
#endif
	int RaysCount = img.h*img.w*kFilter;
	Ray * allRays = (Ray *)malloc (sizeof(Ray)*RaysCount);
	int iii = 0;
//#pragma omp parallel for schedule(dynamic, 1) private(r)       // OpenMP 
	for (unsigned int y = 0; y < img.h; ++y) // Loop over image rows 
	{
		//fprintf (stderr, "\rRendering (%d spp) %5.2f%%", img.samps * 4, 100.*y / (img.h - 1));
		for (unsigned int x = 0; x < img.w; ++x)   // Loop cols 
		{
			int i = (img.h - y - 1)*img.w + x;
#ifdef sampleInCenter
			double sx = 0.5, sy = 0.5;
			double k = 1;
#else
			double k = .25;
			for (unsigned int sy = 0; sy < 2; ++sy) // 2x2 subpixel rows 
			for (unsigned int sx = 0; sx < 2; ++sx) // 2x2 subpixel cols	
#endif
			{
				r = Vec ();
				for (unsigned int sa = 0; sa < img.samps; ++sa)//quality of image
				{
#ifdef filter
					double r1 = 2 * (double)rand () / (double)RAND_MAX, dx = r1 < 1 ? sqrt (r1) - 1 : 1 - sqrt (2 - r1);
					double r2 = 2 * (double)rand () / (double)RAND_MAX, dy = r2 < 1 ? sqrt (r2) - 1 : 1 - sqrt (2 - r2);
#else
					double dx = 0, dy = 0;
#endif
					Vec d = cam.cameraXangle*(((sx + .5 + dx) / 2 + x) / img.w - .5) +
							cam.cameraYangle*(((sy + .5 + dy) / 2 + y) / img.h - .5) +
							cam.cameraLocation.d;
					allRays[i] = Ray (cam.cameraLocation.o + d * 140, d.normalization ());
					i++;
				}
			}

		}
	}
	return allRays;
}
void renderNEW (const Ray * allRays, const  world & wrld, const imgSettings & img, Vec *c)
{
	Vec r;
	for (int i = 0; i < img.h*img.w; i++)
	{
		fprintf (stderr, "\rRendering (%d spp) %5.2f%%", img.samps * 4, (100.*i )/ (img.h*img.w));
#ifdef sampleInCenter
		for (unsigned int sa = 0; sa < img.samps; ++sa)//quality of image
		{
			r = r + RayTrace (wrld, allRays[i], 5)*(1. / img.samps);
		}
#else
		for (unsigned int sa = 0; sa < img.samps; ++sa)//quality of image
		{
			r = r + RayTrace (wrld, allRays[i*4], 5)*(1. / img.samps);
			r = r + RayTrace (wrld, allRays[i*4+1], 5)*(1. / img.samps);
			r = r + RayTrace (wrld, allRays[i*4+2], 5)*(1. / img.samps);
			r = r + RayTrace (wrld, allRays[i*4+3], 5)*(1. / img.samps);
		}
		r = r*0.25;
#endif
		

		c[i] = c[i] + Vec (clamp (r.x), clamp (r.y), clamp (r.z));
	}
}
inline void Render (const  world & wrld, const camera & cam, Vec *c, const imgSettings & img)
{
	Vec r; int i = 0;
#pragma omp parallel for schedule(dynamic, 1) private(r)       // OpenMP 
	for (unsigned int y = 0; y < img.h; ++y) // Loop over image rows 
	{
		fprintf (stderr, "\rRendering (%d spp) %5.2f%%", img.samps * 4, 100.*y / (img.h - 1));
		for (unsigned int x = 0 /*,Xi[3] = { 0, 0, y*y*y }*/; x < img.w; ++x)   // Loop cols 
		{
			//int i = (img.h - y - 1)*img.w + x;
#ifdef sampleInCenter
			double sx = 0.5, sy = 0.5;
			double k = 1;				
#else
			double k = .25;
			for (unsigned int sy = 0; sy < 2; ++sy)     // 2x2 subpixel rows 
			for (unsigned int sx = 0; sx < 2; ++sx) // 2x2 subpixel cols	
#endif
			{
				r = Vec ();
				for (unsigned int sa = 0; sa < img.samps; ++sa)//quality of image
				{
#ifdef filter
					double r1 = 2 * (double)rand () / (double)RAND_MAX, dx = r1 < 1 ? sqrt (r1) - 1 : 1 - sqrt (2 - r1);
					double r2 = 2 * (double)rand () / (double)RAND_MAX, dy = r2 < 1 ? sqrt (r2) - 1 : 1 - sqrt (2 - r2);
#else
					double dx = 0, dy = 0;
#endif
					Vec d = cam.cameraXangle*(((sx + .5 + dx) / 2 + x) / img.w - .5) +
						cam.cameraYangle*(((sy + .5 + dy) / 2 + y) / img.h - .5) +
						cam.cameraLocation.d;
					r = r + RayTrace (wrld, Ray (cam.cameraLocation.o + d * 140, d.normalization ()), 5)*(1. / img.samps);
				}
				c[i] = c[i] + Vec (clamp (r.x), clamp (r.y), clamp (r.z))*k;
				
			}
			i++;
		}
	}
}
int main (int argc, char *argv[])
{
	//int vec = sizeof(Vec);
	//int tr = sizeof(triangle);
	int w = 1024, h = 768, samps = argc == 2 ? atoi (argv[1]) / 4 : 1; // # samples 

	//Ray cam (Vec (85, 45, 170), Vec (-0.45, -0.04, -1).normalization ()); // cam pos, dir 
	camera cam (Ray (Vec (85, 45, 170), Vec (-0.45, -0.04, -1).normalization ()), Vec (w*.5135 / h));
	Vec  r, *c = new Vec[w*h];//cx = Vec (w*.5135 / h), cy = (cx%cam.d).normalization ()*.5135,
	imgSettings img = imgSettings (w, h, samps);
	/*cudaDeviceProp  prop ;
	 cudaGetDeviceProperties (&prop,0);
	 dim3 GRID= dim3 (1024, 1024, 1024);
	 dim3 THREADS= dim3 (512, 512, 64);
	 cudaThreadSynchronize ();*/

	int objCount = sizeof(obj) / sizeof(triangle);


	//memccpy (&crazy [ count], obj, objCount, sizeof(triangle));
	//int objCount = sizeof(obj) / sizeof(triangle);
	int lightsCount = sizeof(lights) / sizeof(Vec);
	world wrld = world (objCount, lightsCount, obj,lights);
	
	
	
	//�������� ������ ��� device ����� ��� OBJECTS
	int size = objCount*sizeof(triangle);
	hipMalloc ((void**)&CUDA_objects, size);
	// �������� ���� �� device
	hipMemcpy (CUDA_objects, &obj, size, hipMemcpyHostToDevice);
	//Ray * allRays = RenderRayHelper (cam, img);
	double start = omp_get_wtime ();
	
	//renderNEW (allRays, wrld, img, c);
	Render/* <<<GRID, THREADS >>> */ (wrld, cam, c, img);
	double end = omp_get_wtime ();

	//free (allRays);
	hipFree (CUDA_objects);
	//cudaThreadSynchronize ();
	fprintf (stderr, "\rtime %5.2f%%", end-start);

	fileOutput (c, w, h);
}